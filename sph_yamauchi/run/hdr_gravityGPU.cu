#include "hip/hip_runtime.h"
#include <particle_simulator.hpp>
#include <hip/hip_runtime.h>
#include <omp.h>
#include "cuda_pointer.h"
#include <hip/hip_runtime_api.h>
#include "DF.hpp"
#include "use_gpu.hpp"
/*
  enum{
  N_THREAD_GPU = 32,
  N_WALK_LIMIT = 1000,
  NI_LIMIT = N_WALK_LIMIT*1000,
  NJ_LIMIT = N_WALK_LIMIT*10000,
  };
*/
class GravityEPI{
public:
  PS::F64vec pos;
  PS::F64 eps2;
};

class GravityEPJ{
public:
  PS::F64 mass;
  PS::F64vec pos;
  PS::F64 eps2;
};

class PS::GravitySPJ{
public:
  PS::F64 mass;
  PS::F64vec pos;
  PS::F64 eps2;
};

class Gravity{
public:
  PS::F64vec acc;
  PS::F64 pot;
  PS::F64 eta;
};

struct Epi_GravityGPU{
  DF3 pos;
  DF eps2;
  int id_walk;
};

struct Epj_GravityGPU{
  DF mass;
  DF3 pos;
  DF eps2;
};

struct Spj_GravityGPU{
  DF mass;
  DF3 pos;
  DF eps2;
};

struct GravityGPU{
  DF3 acc;
  DF pot;
  DF eta;
};

inline __device__ void GravityForce_calc(const struct Epi_GravityGPU ip,
					 const struct Epj_GravityGPU jp,
					 struct GravityGPU *gravity)
{
  DF dpx_ij = ip.pos.x - jp.pos.x;
  DF dpy_ij = ip.pos.y - jp.pos.y;
  DF dpz_ij = ip.pos.z - jp.pos.z;

  DF r2_ij = dpx_ij * dpx_ij + dpy_ij * dpy_ij + dpz_ij * dpz_ij;

  DF re2_i = r2_ij + ip.eps2;
  //DF rei_i = 1. / sqrt(re2_i);
  DF rei_i = rsqrtf(re2_i);
  DF rei3_i = rei_i * rei_i * rei_i;

  DF re2_j = r2_ij + jp.eps2;
  //DF rei_j = 1. / sqrt(re2_j);
  DF rei_j = rsqrtf(re2_j);
  DF rei3_j = rei_j * rei_j * rei_j;
  
  DF dg2_ij = jp.mass * (rei3_i + rei3_j);

  gravity->pot -= jp.mass * (rei_i  + rei_j);
  gravity->acc.x -= dpx_ij * dg2_ij;
  gravity->acc.y -= dpy_ij * dg2_ij;
  gravity->acc.z -= dpz_ij * dg2_ij;
  gravity->eta += jp.mass * rei3_i;

}

__device__ void GravityForceKernel_1walk(struct Epj_GravityGPU *jpsh,
					 const struct Epi_GravityGPU ip,
					 const int id_walk,
					 const int2 *ij_disp,
					 const Epj_GravityGPU *epj,
					 struct GravityGPU *gravity)
{
  const int tid = threadIdx.x;
  const int j_head = ij_disp[id_walk  ].y;
  const int j_tail = ij_disp[id_walk+1].y;

  for(int j=j_head; j<j_tail; j+=N_THREAD_GPU){
    jpsh[tid] = ((Epj_GravityGPU *)(epj + j)) [tid];
   
    if(j_tail-j < N_THREAD_GPU){
      for(int jj=0; jj<j_tail-j; jj++){
	GravityForce_calc(ip, jpsh[jj], gravity);
      }
    }else{
#pragma unroll
      for(int jj=0; jj<N_THREAD_GPU; jj++){
	GravityForce_calc(ip, jpsh[jj], gravity);
      }
    }
  }
	
  gravity->acc.x *= 0.5;
  gravity->acc.y *= 0.5;
  gravity->acc.z *= 0.5;
  gravity->pot *= 0.5;

}

__device__ void GravityForceKernel_2walk(struct Epj_GravityGPU (*jpsh)[N_THREAD_GPU],
					 const struct Epi_GravityGPU ip,
					 const int id_walk,
					 const int2 *ij_disp,
					 const Epj_GravityGPU *epj,
					 struct GravityGPU *gravity,
					 const int iwalk0,
					 const int iwalk1)
{
  const int jbeg0 = ij_disp[iwalk0].y;
  const int jbeg1 = ij_disp[iwalk1].y;
  const int jend0 = ij_disp[iwalk0+1].y;
  const int jend1 = ij_disp[iwalk1+1].y;
  const int nj0 = jend0 - jbeg0;
  const int nj1 = jend1 - jbeg1;

  const int nj_longer = nj0 > nj1 ? nj0 : nj1;
  const int nj_shorter = nj0 > nj1 ? nj1 : nj0;
  const int walk_longer = nj0 > nj1 ? 0 : 1;
  const int jbeg_longer = nj0 > nj1 ? jbeg0 : jbeg1;

  const int mywalk = id_walk==iwalk0 ? 0 : 1;

  const int tid = threadIdx.x;

  for(int j=0; j<nj_shorter; j+=N_THREAD_GPU){
    jpsh[0][tid] = ((Epj_GravityGPU *)(epj + jbeg0 + j))[tid];
    jpsh[1][tid] = ((Epj_GravityGPU *)(epj + jbeg1 + j))[tid];
    if(nj_shorter-j < N_THREAD_GPU){
      for(int jj=0; jj<nj_shorter-j; jj++){	
	GravityForce_calc(ip, jpsh[mywalk][jj], gravity);
      }
    } else{
#pragma unroll
      for(int jj=0; jj<N_THREAD_GPU; jj++){
	GravityForce_calc(ip, jpsh[mywalk][jj], gravity);
      }
    }
  }

  for(int j=nj_shorter; j<nj_longer; j+=N_THREAD_GPU){
    jpsh[0][tid] = ((Epj_GravityGPU *)(epj + jbeg_longer + j))[tid];
    int jrem = nj_longer - j;
    if(jrem < N_THREAD_GPU){
      for(int jj=0; jj<jrem; jj++){
	if(mywalk == walk_longer)
	  GravityForce_calc(ip, jpsh[0][jj], gravity);
      }
    } else{
#pragma unroll
      for(int jj=0; jj<N_THREAD_GPU; jj++){
	if(mywalk == walk_longer)
	  GravityForce_calc(ip, jpsh[0][jj], gravity);
      }
    }
  }

  gravity->acc.x *= 0.5;
  gravity->acc.y *= 0.5;
  gravity->acc.z *= 0.5;
  gravity->pot *= 0.5;
}

struct Walk {
  int nj;
  int jbeg;
  int nwalk;
  int id_walk;
};

inline __device__ void swap(struct Walk *walk_x, struct Walk *walk_y)
{
  struct Walk temp = *walk_x;
  *walk_x = *walk_y;
  *walk_y = temp;
}

__device__ void GravityForceKernel_3walk(struct Epj_GravityGPU (*jpsh)[N_THREAD_GPU],
					 const struct Epi_GravityGPU ip,
					 const int id_walk,
					 const int2 *ij_disp,
					 const Epj_GravityGPU *epj,
					 struct GravityGPU *gravity,
					 const int iwalk0,
					 const int iwalk1,
					 const int iwalk2)
{
  struct Walk walk[3];
  walk[0].jbeg  = ij_disp[iwalk0].y;
  walk[1].jbeg  = ij_disp[iwalk1].y;
  walk[2].jbeg  = ij_disp[iwalk2].y;
  walk[0].nj = ij_disp[iwalk0+1].y - walk[0].jbeg;
  walk[1].nj = ij_disp[iwalk1+1].y - walk[1].jbeg;
  walk[2].nj = ij_disp[iwalk2+1].y - walk[2].jbeg;
  walk[0].nwalk = 0;
  walk[1].nwalk = 1;
  walk[2].nwalk = 2;
  walk[0].id_walk = iwalk0;
  walk[1].id_walk = iwalk1;
  walk[2].id_walk = iwalk2;

  if(walk[0].nj > walk[1].nj) swap(&walk[0], &walk[1]);
  if(walk[1].nj > walk[2].nj) swap(&walk[1], &walk[2]);
  if(walk[0].nj > walk[1].nj) swap(&walk[0], &walk[1]);

  const int nj_longer = walk[2].nj;
  const int nj_middle = walk[1].nj;
  const int nj_shorter = walk[0].nj;

  const int walk_longer = walk[2].nwalk;
  // const int walk_middle = walk[1].nwalk;
  const int walk_shorter = walk[0].nwalk;

  const int jbeg_longer = walk[2].jbeg;
  const int jbeg_middle = walk[1].jbeg;

  const int mywalk = id_walk - iwalk0; 

  // printf("%d %d %d %d %d %d\n", walk_longer, walk_middle, walk_shorter, jend0 - jbeg0, jend1 - jbeg1, jend2 - jbeg2);
  const int tid = threadIdx.x;

  for(int j=0; j<nj_shorter; j+=N_THREAD_GPU){
    jpsh[0][tid] = ((Epj_GravityGPU *)(epj + walk[0].jbeg + j))[tid];
    jpsh[1][tid] = ((Epj_GravityGPU *)(epj + walk[1].jbeg + j))[tid];
    jpsh[2][tid] = ((Epj_GravityGPU *)(epj + walk[2].jbeg + j))[tid];
    if(nj_shorter-j < N_THREAD_GPU){
      for(int jj=0; jj<nj_shorter-j; jj++){	
	GravityForce_calc(ip, jpsh[mywalk][jj], gravity);
      }
    } else{
#pragma unroll
      for(int jj=0; jj<N_THREAD_GPU; jj++){
	GravityForce_calc(ip, jpsh[mywalk][jj], gravity);
      }
    }
  }

  for(int j = nj_shorter; j < nj_middle; j += N_THREAD_GPU){
    jpsh[walk[1].nwalk][tid] = ((Epj_GravityGPU *)(epj + jbeg_middle + j))[tid];
    jpsh[walk[2].nwalk][tid] = ((Epj_GravityGPU *)(epj + jbeg_longer + j))[tid];
    int jrem = nj_middle - j;
    if(jrem < N_THREAD_GPU){
      for(int jj=0; jj<jrem; jj++){
	if(mywalk != walk_shorter) {
	  GravityForce_calc(ip, jpsh[mywalk][jj], gravity);
	}
      }
    } else{
#pragma unroll
      for(int jj=0; jj<N_THREAD_GPU; jj++){
	if(mywalk != walk_shorter)
	  GravityForce_calc(ip, jpsh[mywalk][jj], gravity);
      }
    }
  }

  for(int j=nj_middle; j<nj_longer; j+=N_THREAD_GPU){
    jpsh[walk[2].nwalk][tid] = ((Epj_GravityGPU *)(epj + jbeg_longer + j))[tid];
    int jrem = nj_longer - j;
    if(jrem < N_THREAD_GPU){
      for(int jj=0; jj<jrem; jj++){
	if(mywalk == walk_longer)
	  GravityForce_calc(ip, jpsh[walk[2].nwalk][jj], gravity);
      }
    } else{
#pragma unroll
      for(int jj=0; jj<N_THREAD_GPU; jj++){
	if(mywalk == walk_longer)
	  GravityForce_calc(ip, jpsh[walk[2].nwalk][jj], gravity);
      }
    }
  }

  gravity->acc.x *= 0.5;
  gravity->acc.y *= 0.5;
  gravity->acc.z *= 0.5;
  gravity->pot *= 0.5;
}

__device__ void GravityForceKernel_multiwalk(const struct Epi_GravityGPU ip,
					     const int id_walk,
					     const int2 *ij_disp,
					     const Epj_GravityGPU *epj,
					     struct GravityGPU *gravity)
{
  const int j_head = ij_disp[id_walk  ].y;
  const int j_tail = ij_disp[id_walk+1].y;

  for(int j=j_head; j<j_tail; j++){
    const struct Epj_GravityGPU jp = epj[j];
    GravityForce_calc(ip, jp, gravity);
  }

  gravity->acc.x *= 0.5;
  gravity->acc.y *= 0.5;
  gravity->acc.z *= 0.5;
  gravity->pot *= 0.5;
  
}

__global__ void GravityForceKernel(const int2 *ij_disp,
				   const Epi_GravityGPU *epi,
				   const Epj_GravityGPU *epj,
				   GravityGPU *dev_gravity)
{
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  int t_head = blockDim.x * blockIdx.x;
  int t_tail = t_head + N_THREAD_GPU - 1;
  int nwalk_in_block = 1 + (epi[t_tail].id_walk - epi[t_head].id_walk);
  const int id_walk = epi[tid].id_walk;

  const struct Epi_GravityGPU ip = epi[tid];
  struct GravityGPU gravity;
#ifdef USE_FLOAT
  gravity.acc = make_float3(0.f, 0.f, 0.f);
#endif
#ifdef USE_DOUBLE
  gravity.acc = make_double3(0.f, 0.f, 0.f);
#endif
  gravity.pot = 0.f;
  gravity.eta = 0.f;

  __shared__ struct Epj_GravityGPU jpsh[3][N_THREAD_GPU];
      
  if(1 == nwalk_in_block) {
    GravityForceKernel_1walk(jpsh[0], ip, id_walk, ij_disp, epj, &gravity);
  } else if(2 == nwalk_in_block) {
    const int iwalk0 = epi[t_head].id_walk;
    const int iwalk1 = epi[t_tail].id_walk;
    GravityForceKernel_2walk(jpsh, ip, id_walk, ij_disp, epj, &gravity, iwalk0, iwalk1);
  } else if(3 == nwalk_in_block) {
    // const int iwalk0 = epi[t_head].id_walk;
    // const int iwalk1 = iwalk0 + 1;
    // const int iwalk2 = iwalk0 + 2;
    // GravityForceKernel_3walk(jpsh, ip, id_walk, ij_disp, epj, &gravity, iwalk0, iwalk1, iwalk2);
    GravityForceKernel_multiwalk(ip, id_walk, ij_disp, epj, &gravity);
  } else {
    GravityForceKernel_multiwalk(ip, id_walk, ij_disp, epj, &gravity);
  }
  
  //GravityForceKernel_multiwalk(ip, id_walk, ij_disp, epj, &gravity);
  
  dev_gravity[tid] = gravity;

}

static cudaPointer<Epi_GravityGPU>   dev_epi;
static cudaPointer<Epj_GravityGPU>   dev_epj;
static cudaPointer<Spj_GravityGPU>   dev_spj;
static cudaPointer<GravityGPU> dev_gravity;
static cudaPointer<int2>     ij_disp;
static bool init_call = true;

PS::S32 DispatchKernel_Gravity(const PS::S32 tag,
			       const PS::S32 n_walk,
			       const GravityEPI **epi,
			       const PS::S32 *nip,
			       const GravityEPJ **epj,
			       const PS::S32 *njp,
			       const PS::GravitySPJ **spj,
			       const PS::S32 *nsp)
{
  assert(n_walk <= N_WALK_LIMIT);

  if(init_call){
    dev_epi    .allocate(NI_LIMIT);
    dev_epj    .allocate(NJ_LIMIT);
    dev_gravity.allocate(NI_LIMIT);
    ij_disp    .allocate(N_WALK_LIMIT+2);
    init_call = false;
  }

  ij_disp[0].x = 0;
  ij_disp[0].y = 0;
  for(int k=0; k<n_walk; k++){
    ij_disp[k+1].x = ij_disp[k].x + nip[k];
    ij_disp[k+1].y = ij_disp[k].y + (njp[k] + nsp[k]);
  }
  ij_disp[n_walk+1] = ij_disp[n_walk];

  assert(ij_disp[n_walk].x < NI_LIMIT);
  assert(ij_disp[n_walk].y < NJ_LIMIT);
  ij_disp.htod(n_walk+2);

  int ni_tot_reg = ij_disp[n_walk].x;
  if(ni_tot_reg % N_THREAD_GPU){
    ni_tot_reg /= N_THREAD_GPU;
    ni_tot_reg++;
    ni_tot_reg *= N_THREAD_GPU;
  }

  int ni_tot = 0;
  int nj_tot = 0;
  
  int iw, i, j, num_i, num_j, num_sj;
#pragma omp parallel for private(i,j, num_i, num_j, num_sj) reduction(+:ni_tot, nj_tot)
  for(iw=0; iw<n_walk; iw++){
    for(i=0; i<nip[iw]; i++){
      num_i = ij_disp[iw].x + i;
      dev_epi[num_i].pos.x = epi[iw][i].pos.x;
      dev_epi[num_i].pos.y = epi[iw][i].pos.y;
      dev_epi[num_i].pos.z = epi[iw][i].pos.z;
      dev_epi[num_i].eps2 = epi[iw][i].eps2;
      dev_epi[num_i].id_walk = iw;
      ni_tot++;
    }
    for(j=0; j<njp[iw]; j++){
      num_j = ij_disp[iw].y + j;
      dev_epj[num_j].pos.x = epj[iw][j].pos.x;
      dev_epj[num_j].pos.y = epj[iw][j].pos.y;
      dev_epj[num_j].pos.z = epj[iw][j].pos.z;
      dev_epj[num_j].mass = epj[iw][j].mass;
      dev_epj[num_j].eps2 = epj[iw][j].eps2;
      nj_tot++;
    }
    for(j=0; j<nsp[iw]; j++){
      num_sj = ij_disp[iw].y + njp[iw] + j;
      dev_epj[num_sj].pos.x = spj[iw][j].pos.x;
      dev_epj[num_sj].pos.y = spj[iw][j].pos.y;
      dev_epj[num_sj].pos.z = spj[iw][j].pos.z;
      dev_epj[num_sj].mass = spj[iw][j].mass;
      dev_epj[num_sj].eps2 = spj[iw][j].eps2;
      nj_tot++;
    }
  }

  for(int i=ni_tot; i<ni_tot_reg; i++){
    dev_epi[i].id_walk = n_walk;
  }

  dev_epi.htod(ni_tot_reg);
  dev_epj.htod(nj_tot);

  int nblocks  = ni_tot_reg / N_THREAD_GPU;
  int nthreads = N_THREAD_GPU;

  GravityForceKernel <<<nblocks, nthreads>>> (ij_disp, dev_epi, dev_epj, dev_gravity);

  return 0;
}

PS::S32 RetrieveKernel_Gravity(const PS::S32 tag,
			       const PS::S32 n_walk,
			       const PS::S32 *ni,
			       Gravity **gravity)
{
  int ni_tot = 0;
  for(int k=0; k<n_walk; k++){
    ni_tot += ni[k];
  }
  dev_gravity.dtoh(ni_tot);

  int n_cnt = 0;
  for(int iw=0; iw<n_walk; iw++){
    for(int i=0; i<ni[iw]; i++){ 
      gravity[iw][i].acc.x = dev_gravity[n_cnt].acc.x;
      gravity[iw][i].acc.y = dev_gravity[n_cnt].acc.y;
      gravity[iw][i].acc.z = dev_gravity[n_cnt].acc.z;
      gravity[iw][i].pot = dev_gravity[n_cnt].pot;
      gravity[iw][i].eta = dev_gravity[n_cnt].eta;
      n_cnt++;
    }
  }

  return 0;
}
