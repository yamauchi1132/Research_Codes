#include "hip/hip_runtime.h"
#include <particle_simulator.hpp>
#include <omp.h>
#include "cuda_pointer.h"
#include "hip/hip_runtime_api.h"
#include "DF.hpp"
#include "use_gpu.hpp"

__constant__ PS::F64 KernelSupportRadiusMaximum_C;
__constant__ DF eta_c;
__constant__ DF ksrh_c;
__constant__ DF ceff0_c;
__constant__ DF ceff1_c;
/*
  enum{
  N_THREAD_GPU = 32,
  N_WALK_LIMIT = 1000,
  NI_LIMIT = N_WALK_LIMIT*1000,
  NJ_LIMIT = N_WALK_LIMIT*10000,
  NumberOfDimension = 3
  };
*/
class DensityEPI
{
public:
  PS::S32    id;
  PS::F64    mass;
  PS::F64vec pos;
  PS::F64vec vel;
  PS::F64    ksr;
  PS::F64    rs;
};

class DensityEPJ
{
public:
  PS::S32    id;
  PS::F64    mass;
  PS::F64vec pos;
  PS::F64vec vel;
};

class Density
{
public:
  PS::F64 dens;
  PS::F64 divv;
  PS::F64 rotv;
  PS::F64 grdh;
  PS::F64 ksr;
  PS::S64 np;
  bool    itr;
};

struct Epi_DensGPU{
  long long int id;
  DF mass;
  DF3 pos;
  DF3 vel;
  DF ksr;
  DF rs;
  int id_walk;
  int nocalc;
};

struct Epj_DensGPU{
  long long int id;
  DF mass;
  DF3 pos;
  DF3 vel;
};

struct DensGPU{
  DF dens;
  DF divv;
  DF rotv;
  DF grdh;
  DF ksr;
  long long int np;
  bool itr;
};

inline __device__ DF kernel0th(const DF q)
{
  //const DF ceff0   = +3.342253804929802286e+00;
  const DF qmin  = ((1. - q > 0.) ? 1. - q : 0.);
  const DF qmin2 = qmin * qmin;
  return ceff0_c * qmin2 * qmin2 * (1. + 4. * q);
}

inline __device__ DF kernel1st(const DF q)
{
  //const DF ceff1 = +1.336901521971920914e+01;
  const DF qmin  = ((1. - q > 0.) ? 1. - q : 0.);
  const DF qmin2 = qmin  * qmin;
  const DF qmin3 = qmin  * qmin2;
  const DF qmin4 = qmin2 * qmin2;
  return ceff1_c * (qmin4 - qmin3 * (1. + 4. * q));
}

inline __device__ void Dens_calc1(const struct Epi_DensGPU ip,
				  struct Epj_DensGPU jp,
				  const DF hi_i,
				  const DF hi3_i,
				  struct DensGPU *dens)
{
  const DF dx_ij = ip.pos.x - jp.pos.x;
  const DF dy_ij = ip.pos.y - jp.pos.y;
  const DF dz_ij = ip.pos.z - jp.pos.z;

  const DF r2_ij = dx_ij * dx_ij + dy_ij * dy_ij + dz_ij * dz_ij;

  const DF r1_ij = sqrt(r2_ij);
  const DF q_i   = r1_ij * hi_i;

  const DF kw0 = kernel0th(q_i);
  const DF rhj = jp.mass * hi3_i * kw0;
  
  dens->dens += rhj;
  dens->np += ((q_i < 1.) ? 1 : 0);        
}

inline __device__ void Dens_calc2(struct Epi_DensGPU ip,
				  const struct Epj_DensGPU jp,
				  const DF hi_i,
				  const DF hi4_i,
				  DF *grdh_i,
				  DF *divv_i,
				  DF *rotx_i,
				  DF *roty_i,
				  DF *rotz_i)
{
  DF dpx_ij = ip.pos.x - jp.pos.x;
  DF dpy_ij = ip.pos.y - jp.pos.y;
  DF dpz_ij = ip.pos.z - jp.pos.z;
  DF dvx_ij = ip.vel.x - jp.vel.x;
  DF dvy_ij = ip.vel.y - jp.vel.y;
  DF dvz_ij = ip.vel.z - jp.vel.z;

  DF r2_ij = dpx_ij * dpx_ij + dpy_ij * dpy_ij + dpz_ij * dpz_ij;
  //DF ri_ij = 1. / sqrt(r2_ij);
  DF ri_ij = rsqrtf(r2_ij);
  ri_ij = ((ip.id != jp.id) ? ri_ij : 0.);
  DF r1_ij = r2_ij * ri_ij;
  DF q_i = r1_ij * hi_i;

  DF kw0 = kernel0th(q_i);
  DF kw1 = kernel1st(q_i);

  DF m_j(jp.mass);
  DF ghj = (DF) NumberOfDimension * kw0;
	   
  ghj += q_i * kw1;
  *grdh_i -= ghj * hi4_i * m_j;

  DF dw_ij  = m_j * hi4_i * kw1 * ri_ij;
  DF dwx_ij = dw_ij * dpx_ij;
  DF dwy_ij = dw_ij * dpy_ij;
  DF dwz_ij = dw_ij * dpz_ij;

  *divv_i -= dvx_ij * dwx_ij;
  *divv_i -= dvy_ij * dwy_ij;
  *divv_i -= dvz_ij * dwz_ij;

  *rotx_i += dvy_ij * dwz_ij;
  *roty_i += dvz_ij * dwx_ij;
  *rotz_i += dvx_ij * dwy_ij;
  *rotx_i -= dvz_ij * dwy_ij;
  *roty_i -= dvx_ij * dwz_ij;
  *rotz_i -= dvy_ij * dwx_ij;  

}

__device__ void DensKernel_1walk(struct Epj_DensGPU *jpsh,
				 struct Epi_DensGPU ip,
				 const int id_walk,
				 const int2 *ij_disp,
				 const Epj_DensGPU *epj,
				 struct DensGPU *dens)
{
  const int tid = threadIdx.x;
  const int j_head = ij_disp[id_walk  ].y;
  const int j_tail = ij_disp[id_walk+1].y;

  for(int repeat=0; repeat<3; repeat++){
    const DF hi_i  = 1. / ip.ksr;
    const DF hi3_i = hi_i * hi_i * hi_i;
    //const DF hi4_i = hi_i * hi3_i;
    dens->dens = 0;
    dens->np = 0;
    //DF rh_i  = 0.;
    //long long int nj_i  = 0.;
    for(int j=j_head; j<j_tail; j+=N_THREAD_GPU){
      jpsh[tid] = ((Epj_DensGPU *)(epj + j))[tid];

      if(j_tail-j < N_THREAD_GPU){
	for(int jj=0; jj<j_tail-j; jj++){
	  Dens_calc1(ip, jpsh[jj], hi_i, hi3_i, dens);
	}
      } else {
#pragma unroll
	for(int jj=0; jj<N_THREAD_GPU; jj++){
	  Dens_calc1(ip, jpsh[jj], hi_i, hi3_i, dens);
	}
      }
    }
    /*
      const DF eta     = 1.6;
      const DF ksrh    = 1.936492;
    */
    DF buf_hs = eta_c * ksrh_c * powf(ip.mass / dens->dens, 1. / 3.);
  
    buf_hs = ((buf_hs < KernelSupportRadiusMaximum_C)
	      ? buf_hs : KernelSupportRadiusMaximum_C);

    dens->ksr = buf_hs;
    dens->itr = (buf_hs > ip.rs) ? true : false;
 
    ip.ksr = buf_hs;
  }

  const DF hi_i   = 1. / ip.ksr;
  const DF hi4_i  = hi_i * hi_i * hi_i * hi_i;
  DF grdh_i = 0.;
  DF divv_i = 0.;
  DF rotx_i = 0.;
  DF roty_i = 0.;
  DF rotz_i = 0.;

  for(int j=j_head; j<j_tail; j+=N_THREAD_GPU){
    jpsh[tid] = ((Epj_DensGPU *)(epj + j))[tid];
    if(j_tail-j < N_THREAD_GPU){
      for(int jj=0; jj<j_tail-j; jj++){
	Dens_calc2(ip, jpsh[jj], hi_i, hi4_i, &grdh_i, &divv_i, &rotx_i, &roty_i, &rotz_i); 
      }
    } else {
#pragma unroll
      for(int jj=0; jj<N_THREAD_GPU; jj++){
	Dens_calc2(ip, jpsh[jj], hi_i, hi4_i, &grdh_i, &divv_i, &rotx_i, &roty_i, &rotz_i); 
      }
    }
  }

  DF dens_i = dens->dens;
  DF deni_i = 1. / dens_i;
  DF omgi_i = 1. / (1. + ip.ksr * deni_i * grdh_i / NumberOfDimension);
  omgi_i = (1. + ip.ksr * deni_i * grdh_i / NumberOfDimension != 0.) ? omgi_i : 1.;
  DF rot2_i = rotx_i * rotx_i + roty_i * roty_i + rotz_i * rotz_i;
  DF rotv_i = rot2_i * ((rot2_i != 0.) ? 1. / sqrt(rot2_i) : 0.);
  rotv_i *= deni_i * omgi_i;
  divv_i *= deni_i * omgi_i;

  dens->divv = divv_i;
  dens->rotv = rotv_i;
  dens->grdh = omgi_i;
}

__device__ void DensKernel_2walk(struct Epj_DensGPU (*jpsh)[N_THREAD_GPU],
				 struct Epi_DensGPU ip,
				 const int id_walk,
				 const int2 *ij_disp,
				 const Epj_DensGPU *epj,
				 struct DensGPU *dens,
				 const int iwalk0,
				 const int iwalk1)
{
  const int jbeg0 = ij_disp[iwalk0].y;
  const int jbeg1 = ij_disp[iwalk1].y;
  const int jend0 = ij_disp[iwalk0+1].y;
  const int jend1 = ij_disp[iwalk1+1].y;
  const int nj0 = jend0 - jbeg0;
  const int nj1 = jend1 - jbeg1;

  const int nj_longer = nj0 > nj1 ? nj0 : nj1;
  const int nj_shorter = nj0 > nj1 ? nj1 : nj0;
  const int walk_longer = nj0 > nj1 ? 0 : 1;
  const int jbeg_longer = nj0 > nj1 ? jbeg0 : jbeg1;

  const int mywalk = id_walk==iwalk0 ? 0 : 1;

  const int tid = threadIdx.x;

  for(int repeat=0; repeat<3; repeat++){
    const DF hi_i  = 1. / ip.ksr;
    const DF hi3_i = hi_i * hi_i * hi_i;
    //const DF hi4_i = hi_i * hi3_i;
    dens->dens = 0;
    dens->np = 0;
    //DF rh_i  = 0.;
    //long long int nj_i  = 0.;
    for(int j=0; j<nj_shorter; j+=N_THREAD_GPU){
      jpsh[0][tid] = ((Epj_DensGPU *)(epj + jbeg0 + j))[tid];
      jpsh[1][tid] = ((Epj_DensGPU *)(epj + jbeg1 + j))[tid];
      if(nj_shorter-j < N_THREAD_GPU){
	for(int jj=0; jj<nj_shorter-j; jj++){
	  Dens_calc1(ip, jpsh[mywalk][jj], hi_i, hi3_i, dens);
	}
      }else {
#pragma unroll
	for(int jj=0; jj<N_THREAD_GPU; jj++){
	  Dens_calc1(ip, jpsh[mywalk][jj], hi_i, hi3_i, dens);
	}
      }
    }

    for(int j=nj_shorter; j<nj_longer; j+=N_THREAD_GPU){
      jpsh[0][tid] = ((Epj_DensGPU *)(epj + jbeg_longer + j))[tid];
      int jrem = nj_longer - j;
      if(jrem < N_THREAD_GPU){
	for(int jj=0; jj<jrem; jj++){
	  if(mywalk == walk_longer)
	    Dens_calc1(ip, jpsh[0][jj], hi_i, hi3_i, dens);
	}
      }else {
#pragma unroll
	for(int jj=0; jj<N_THREAD_GPU; jj++){
	  if(mywalk == walk_longer)
	    Dens_calc1(ip, jpsh[0][jj], hi_i, hi3_i, dens); 
	}
      }
    }
    /*
      const DF eta     = 1.6;
      const DF ksrh    = 1.936492;
    */
    DF buf_hs = eta_c * ksrh_c * powf(ip.mass / dens->dens, 1. / 3.);

    buf_hs = ((buf_hs < KernelSupportRadiusMaximum_C)
	      ? buf_hs : KernelSupportRadiusMaximum_C);

    dens->ksr = buf_hs;
    dens->itr = (buf_hs > ip.rs) ? true : false;
 
    ip.ksr = buf_hs;
  }

  const DF hi_i   = 1. / ip.ksr;
  const DF hi4_i  = hi_i * hi_i * hi_i * hi_i;
  DF grdh_i = 0.;
  DF divv_i = 0.;
  DF rotx_i = 0.;
  DF roty_i = 0.;
  DF rotz_i = 0.;

  for(int j=0; j<nj_shorter; j+=N_THREAD_GPU){
    jpsh[0][tid] = ((Epj_DensGPU *)(epj + jbeg0 + j))[tid];
    jpsh[1][tid] = ((Epj_DensGPU *)(epj + jbeg1 + j))[tid];
    if(nj_shorter-j < N_THREAD_GPU){
      for(int jj=0; jj<nj_shorter-j; jj++){
	Dens_calc2(ip, jpsh[mywalk][jj], hi_i, hi4_i, &grdh_i, &divv_i, &rotx_i, &roty_i, &rotz_i); 
      }
    }else {
#pragma unroll
      for(int jj=0; jj<N_THREAD_GPU; jj++){
	Dens_calc2(ip, jpsh[mywalk][jj], hi_i, hi4_i, &grdh_i, &divv_i, &rotx_i, &roty_i, &rotz_i); 
      }
    }
  }

  for(int j=nj_shorter; j<nj_longer; j+=N_THREAD_GPU){
    jpsh[0][tid] = ((Epj_DensGPU *)(epj + jbeg_longer + j))[tid];
    int jrem = nj_longer - j;
    if(jrem < N_THREAD_GPU){
      for(int jj=0; jj<jrem; jj++){
	if(mywalk == walk_longer)
	  Dens_calc2(ip, jpsh[0][jj], hi_i, hi4_i, &grdh_i, &divv_i, &rotx_i, &roty_i, &rotz_i); 
      }
    }else {
#pragma unroll
      for(int jj=0; jj<N_THREAD_GPU; jj++){
	if(mywalk == walk_longer)
	  Dens_calc2(ip, jpsh[0][jj], hi_i, hi4_i, &grdh_i, &divv_i, &rotx_i, &roty_i, &rotz_i); 
      }
    }
  }

  DF dens_i = dens->dens;
  DF deni_i = 1. / dens_i;
  DF omgi_i = 1. / (1. + ip.ksr * deni_i * grdh_i / NumberOfDimension);
  omgi_i = (1. + ip.ksr * deni_i * grdh_i / NumberOfDimension != 0.) ? omgi_i : 1.;
  DF rot2_i = rotx_i * rotx_i + roty_i * roty_i + rotz_i * rotz_i;
  DF rotv_i = rot2_i * ((rot2_i != 0.) ? 1. / sqrt(rot2_i) : 0.);
  rotv_i *= deni_i * omgi_i;
  divv_i *= deni_i * omgi_i;

  dens->divv = divv_i;
  dens->rotv = rotv_i;
  dens->grdh = omgi_i;
}

__device__ void DensKernel_multiwalk(struct Epi_DensGPU ip,
				     const int id_walk,
				     const int2 *ij_disp,
				     const Epj_DensGPU *epj,
				     struct DensGPU *dens)
{
  const int j_head = ij_disp[id_walk  ].y;
  const int j_tail = ij_disp[id_walk+1].y;
 
  for(int repeat=0; repeat<3; repeat++){
    const DF hi_i  = 1. / ip.ksr;
    const DF hi3_i = hi_i * hi_i * hi_i;
    //const DF hi4_i = hi_i * hi3_i;
    dens->dens = 0;
    dens->np = 0;
    //DF rh_i  = 0.;
    //long long int nj_i  = 0.;
    
    for(int j=j_head; j<j_tail; j++){
      const struct Epj_DensGPU jp = epj[j];
      Dens_calc1(ip, jp, hi_i, hi3_i, dens);
    }
    /*   
	 const DF eta     = 1.6;
	 const DF ksrh    = 1.936492;
    */
    DF buf_hs = eta_c * ksrh_c * powf(ip.mass / dens->dens, 1. / 3.);
  
    buf_hs = ((buf_hs < KernelSupportRadiusMaximum_C)
	      ? buf_hs : KernelSupportRadiusMaximum_C);
   
    dens->ksr = buf_hs;
    dens->itr = (buf_hs > ip.rs) ? true : false;
 
    ip.ksr = buf_hs;
  }
  const DF hi_i   = 1. / ip.ksr;
  const DF hi4_i  = hi_i * hi_i * hi_i * hi_i;
  DF grdh_i = 0.;
  DF divv_i = 0.;
  DF rotx_i = 0.;
  DF roty_i = 0.;
  DF rotz_i = 0.;
  
  for(int j=j_head; j<j_tail; j++){
    const struct Epj_DensGPU jp = epj[j];
    Dens_calc2(ip, jp, hi_i, hi4_i, &grdh_i, &divv_i, &rotx_i, &roty_i, &rotz_i); 
  }
  
  DF dens_i = dens->dens;
  DF deni_i = 1. / dens_i;
  DF omgi_i = 1. / (1. + ip.ksr * deni_i * grdh_i / NumberOfDimension);
  omgi_i = (1. + ip.ksr * deni_i * grdh_i / NumberOfDimension != 0.) ? omgi_i : 1.;
  DF rot2_i = rotx_i * rotx_i + roty_i * roty_i + rotz_i * rotz_i;
  DF rotv_i = rot2_i * ((rot2_i != 0.) ? 1. / sqrt(rot2_i) : 0.);
  rotv_i *= deni_i * omgi_i;
  divv_i *= deni_i * omgi_i;

  dens->divv = divv_i;
  dens->rotv = rotv_i;
  dens->grdh = omgi_i;
  
}

__global__ void DensKernel(const int2 *ij_disp,
			   const Epi_DensGPU *epi,
			   const Epj_DensGPU *epj,
			   DensGPU *dev_dens)  
{
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  struct Epi_DensGPU ip = epi[tid];
  const int id_walk = epi[tid].id_walk;
  struct DensGPU dens;

  const int t_head = blockDim.x * blockIdx.x;
  const int t_tail = t_head + N_THREAD_GPU - 1;
  const int nwalk_in_block = 1 + (epi[t_tail].id_walk - epi[t_head].id_walk);

  __shared__ struct Epj_DensGPU jpsh[2][N_THREAD_GPU];
    
  if(1 == nwalk_in_block){
    DensKernel_1walk(jpsh[0], ip, id_walk, ij_disp, epj, &dens);
  } else if(2 == nwalk_in_block){
    const int iwalk0 = epi[t_head].id_walk;
    const int iwalk1 = epi[t_tail].id_walk;
    DensKernel_2walk(jpsh, ip, id_walk, ij_disp, epj, &dens, iwalk0, iwalk1);
  } else {
    DensKernel_multiwalk(ip, id_walk, ij_disp, epj, &dens);
  }
  
  //DensKernel_multiwalk(ip, id_walk, ij_disp, epj, &dens);

  dev_dens[tid] = dens;

}

static cudaPointer<Epi_DensGPU>   dev_epi;
static cudaPointer<Epj_DensGPU>   dev_epj;
static cudaPointer<DensGPU> dev_dens;
static cudaPointer<int2>     ij_disp;
static bool init_call = true;

namespace RunParameter {
  extern PS::F64 KernelSupportRadiusMaximum;
};

PS::S32 DispatchKernel_Dens(const PS::S32 tag,
			    const PS::S32 n_walk,
			    const DensityEPI **epi,
			    const PS::S32 *n_epi,
			    const DensityEPJ **epj,
			    const PS::S32 *n_epj)
{
  assert(n_walk <= N_WALK_LIMIT);

  if(init_call){
    dev_epi  .allocate(NI_LIMIT);
    dev_epj  .allocate(NJ_LIMIT);
    dev_dens .allocate(NI_LIMIT);
    ij_disp  .allocate(N_WALK_LIMIT+2);
    init_call = false;
  }

  ij_disp[0].x = 0;
  ij_disp[0].y = 0;
  for(int k=0; k<n_walk; k++){
    ij_disp[k+1].x = ij_disp[k].x + n_epi[k];
    ij_disp[k+1].y = ij_disp[k].y + n_epj[k];
  }
  ij_disp[n_walk+1] = ij_disp[n_walk];

  assert(ij_disp[n_walk].x < NI_LIMIT);
  assert(ij_disp[n_walk].y < NJ_LIMIT);
  ij_disp.htod(n_walk+2);

  int ni_tot_reg = ij_disp[n_walk].x;
  if(ni_tot_reg % N_THREAD_GPU){
    ni_tot_reg /= N_THREAD_GPU;
    ni_tot_reg++;
    ni_tot_reg *= N_THREAD_GPU;
  }

  int ni_tot = 0;
  int nj_tot = 0;
  
  int iw, i, j, num_i, num_j;

#pragma omp parallel for private(i,j, num_i, num_j) reduction(+:ni_tot, nj_tot)
  for(iw=0; iw<n_walk; iw++){
    for(i=0; i<n_epi[iw]; i++){
      num_i = ij_disp[iw].x + i;
      dev_epi[num_i].id = epi[iw][i].id;
      dev_epi[num_i].pos.x = epi[iw][i].pos.x;
      dev_epi[num_i].pos.y = epi[iw][i].pos.y;
      dev_epi[num_i].pos.z = epi[iw][i].pos.z;
      dev_epi[num_i].vel.x = epi[iw][i].vel.x;
      dev_epi[num_i].vel.y = epi[iw][i].vel.y;
      dev_epi[num_i].vel.z = epi[iw][i].vel.z;
      dev_epi[num_i].ksr = epi[iw][i].ksr;
      dev_epi[num_i].mass = epi[iw][i].mass;
      dev_epi[num_i].rs = epi[iw][i].rs;
      dev_epi[num_i].id_walk = iw;
      ni_tot++;
    }
    for(j=0; j<n_epj[iw]; j++){
      num_j = ij_disp[iw].y + j;
      dev_epj[num_j].id = epj[iw][j].id;
      dev_epj[num_j].pos.x = epj[iw][j].pos.x;
      dev_epj[num_j].pos.y = epj[iw][j].pos.y;
      dev_epj[num_j].pos.z = epj[iw][j].pos.z;
      dev_epj[num_j].vel.x = epj[iw][j].vel.x;
      dev_epj[num_j].vel.y = epj[iw][j].vel.y;
      dev_epj[num_j].vel.z = epj[iw][j].vel.z;
      dev_epj[num_j].mass = epj[iw][j].mass;
      nj_tot++;
    }
  }

  for(int i=ni_tot; i<ni_tot_reg; i++){
    dev_epi[i].id_walk = n_walk;
    dev_epi[i].nocalc = 1;
  }

  dev_epi.htod(ni_tot_reg);
  dev_epj.htod(nj_tot);

  const DF eta = 1.6;
  const DF ksrh = 1.936492;
  const DF ceff0   = +3.342253804929802286e+00;
  const DF ceff1 = +1.336901521971920914e+01;

  hipMemcpyToSymbol(HIP_SYMBOL(KernelSupportRadiusMaximum_C), &RunParameter::KernelSupportRadiusMaximum, sizeof(PS::F64));
  hipMemcpyToSymbol(HIP_SYMBOL(eta_c), &eta, sizeof(DF));
  hipMemcpyToSymbol(HIP_SYMBOL(ksrh_c), &ksrh, sizeof(DF));
  hipMemcpyToSymbol(HIP_SYMBOL(ceff0_c), &ceff0, sizeof(DF));
  hipMemcpyToSymbol(HIP_SYMBOL(ceff1_c), &ceff1, sizeof(DF));

  int nblocks  = ni_tot_reg / N_THREAD_GPU;
  int nthreads = N_THREAD_GPU;
  DensKernel <<<nblocks, nthreads>>> (ij_disp, dev_epi, dev_epj, dev_dens);

  return 0;
}

PS::S32 RetrieveKernel_Dens(const PS::S32 tag,
			    const PS::S32 n_walk,
			    const PS::S32 *ni,
			    Density **density)
{
  int ni_tot = 0;
  for(int k=0; k<n_walk; k++){
    ni_tot += ni[k];
  }

  dev_dens.dtoh(ni_tot);

  int n_cnt = 0;
  for(int iw=0; iw<n_walk; iw++){
    for(int i=0; i<ni[iw]; i++){
      density[iw][i].dens = dev_dens[n_cnt].dens;
      density[iw][i].divv = dev_dens[n_cnt].divv;
      density[iw][i].rotv = dev_dens[n_cnt].rotv;
      density[iw][i].grdh = dev_dens[n_cnt].grdh;
      density[iw][i].ksr = dev_dens[n_cnt].ksr;
      density[iw][i].np = dev_dens[n_cnt].np;
      density[iw][i].itr = dev_dens[n_cnt].itr;
      n_cnt++;
    }
  }

  return 0;
}
