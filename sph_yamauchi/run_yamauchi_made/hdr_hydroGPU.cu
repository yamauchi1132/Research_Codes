#include "hip/hip_runtime.h"
#include <particle_simulator.hpp>
#include <omp.h>
#include "cuda_pointer.h"
#include "hip/hip_runtime_api.h"
#include "DF.hpp"
#include "use_gpu.hpp"

__constant__ DF ceff1_c;
/*
  enum{
  N_THREAD_GPU = 32,
  N_WALK_LIMIT = 1000,
  NI_LIMIT = N_WALK_LIMIT*1000,
  NJ_LIMIT = N_WALK_LIMIT*10000,
  };
*/
class HydroEPI{
public:
  PS::S64 id;
  PS::F64vec pos;
  PS::F64vec vel;
  PS::F64    ksr;
  PS::F64    hinv;
  PS::F64    pres;
  PS::F64    presc;
  PS::F64    bswt;
  PS::F64    dens;
  PS::F64    vsnd;
  PS::F64    alph;
  PS::F64    alphu;
  PS::F64    thrm;
  PS::F64    eta;
};

class HydroEPJ{
public:
  PS::S64 id;
  PS::F64    mass;
  PS::F64vec pos;
  PS::F64vec vel;
  PS::F64    ksr;
  PS::F64    hinv;
  PS::F64    pres;
  PS::F64    presc;
  PS::F64    bswt;
  PS::F64    dens;
  PS::F64    vsnd;
  PS::F64    alph;
  PS::F64    alphu;
  PS::F64    thrm;
  PS::F64    eta;
};

class Hydro{
public:
  PS::F64vec acch;
  PS::F64vec accg;
  PS::F64    udot;
  PS::F64    vsmx;
  PS::F64    diffu;
};

struct Epi_HydroGPU{
  long long int id;
  DF3 pos;
  DF3 vel;
  DF hinv;
  DF pres;
  DF presc;
  DF bswt;
  DF dens;
  DF vsnd;
  DF alph;
  DF alphu;
  DF thrm;
  DF eta;
  int id_walk;
};

struct Epj_HydroGPU{
  long long int id;
  DF3 pos;
  DF3 vel;
  DF hinv;
  DF pres;
  DF presc;
  DF bswt;
  DF dens;
  DF vsnd;
  DF alph;
  DF alphu;
  DF thrm;
  DF eta;
  DF mass;
};

struct HydroGPU{
  DF3 acc;
  DF udot;
  DF vsmx;
  DF diffu;
  DF3 accg;
};

inline __device__ DF kernel1st(const DF q)
{
  // const DF ceff1 = +1.336901521971920914e+01;
  const DF qmin  = ((1. - q > 0.) ? 1. - q : 0.);
  const DF qmin2 = qmin  * qmin;
  const DF qmin3 = qmin  * qmin2;
  const DF qmin4 = qmin2 * qmin2;
  return ceff1_c * (qmin4 - qmin3 * (1. + 4. * q));
}

inline __device__ void HydroForce_calc(const struct Epi_HydroGPU ip,
				       const struct Epj_HydroGPU jp,
				       DF3 *acc,
				       DF3 *accg,
				       DF *udot,
				       DF *vsmx,
				       DF *diffu,
				       const DF hi4_i)
{
  const DF dpx_ij = ip.pos.x - jp.pos.x;
  const DF dpy_ij = ip.pos.y - jp.pos.y;
  const DF dpz_ij = ip.pos.z - jp.pos.z;
  const DF dvx_ij = ip.vel.x - jp.vel.x;
  const DF dvy_ij = ip.vel.y - jp.vel.y;
  const DF dvz_ij = ip.vel.z - jp.vel.z;

  const DF r2_ij = dpx_ij * dpx_ij + dpy_ij * dpy_ij + dpz_ij * dpz_ij;
  //const DF ri_ij = ((ip.id != jp.id) ? 1. / sqrt(r2_ij) : 0.);
  const DF ri_ij = ((ip.id != jp.id) ? rsqrtf(r2_ij) : 0.);
  const DF r1_ij = r2_ij * ri_ij;
  const DF q_i   = r1_ij * ip.hinv;
  const DF q_j   = r1_ij * jp.hinv;

  const DF hi4_j = jp.hinv * jp.hinv * jp.hinv * jp.hinv;
  const DF dw_i  = hi4_i * kernel1st(q_i);
  const DF dw_j  = hi4_j * kernel1st(q_j);
  const DF ka_ij = (dw_i + dw_j) * jp.mass;

  const DF rv_ij  = dpx_ij * dvx_ij + dpy_ij * dvy_ij + dpz_ij * dvz_ij;
  const DF w_ij   = rv_ij * ri_ij;
  const DF w0_ij  = ((w_ij < 0.) ? w_ij : 0.);
  const DF vs_ij  = ip.vsnd + jp.vsnd - 3. * w0_ij;
  const DF rhi_ij = 1. / (ip.dens + jp.dens);
  const DF av0_ij = (ip.bswt + jp.bswt) * (ip.alph + jp.alph) * vs_ij * w0_ij;
  
  *vsmx  = ((*vsmx > vs_ij) ? *vsmx : vs_ij);

  const DF ta_ij = (ip.presc + jp.presc - 0.5 * av0_ij * rhi_ij) * ka_ij * ri_ij;
  acc->x -= ta_ij * dpx_ij;
  acc->y -= ta_ij * dpy_ij;
  acc->z -= ta_ij * dpz_ij;

  const DF vsu2_ij = fabs(ip.pres - jp.pres) * rhi_ij * 2.;
  const DF vsui_ij = ((vsu2_ij != 0.) ? 1. / sqrt(vsu2_ij) : 0.);
  const DF vsu_ij  = vsu2_ij * vsui_ij;
  const DF du_ij   = ip.thrm - jp.thrm;

  *udot += ka_ij * (ip.presc * w_ij - rhi_ij * (0.25 * av0_ij * w0_ij - (ip.alphu + jp.alphu) * vsu_ij * du_ij));

  *diffu += rhi_ij * du_ij * ka_ij * ri_ij;

  const DF dg_ij = jp.mass * ri_ij * (ip.eta * dw_i + jp.eta * dw_j);
  accg->x += dg_ij * dpx_ij;
  accg->y += dg_ij * dpy_ij;
  accg->z += dg_ij * dpz_ij;
}

__device__ void HydroForceKernel_1walk(struct Epj_HydroGPU *jpsh,
				       const struct Epi_HydroGPU ip,
				       const Epj_HydroGPU *epj,
				       const int id_walk,
				       const int2 *ij_disp,
				       DF3 *acc,
				       DF3 *accg,
				       DF *udot,
				       DF *vsmx,
				       DF *diffu)
{
  const int tid = threadIdx.x;
  const int j_head = ij_disp[id_walk  ].y;
  const int j_tail = ij_disp[id_walk+1].y;

  const DF hi4_i = ip.hinv * ip.hinv * ip.hinv * ip.hinv;

  for(int j=j_head; j<j_tail; j+=N_THREAD_GPU){
    jpsh[tid] = ((Epj_HydroGPU *)(epj + j))[tid];
    if(j_tail-j < N_THREAD_GPU){
      for(int jj=0; jj<j_tail-j; jj++){
	HydroForce_calc(ip, jpsh[jj], acc, accg, udot, vsmx, diffu, hi4_i);
      }
    }else{
#pragma unroll
      for(int jj=0; jj<N_THREAD_GPU; jj++){
	HydroForce_calc(ip, jpsh[jj], acc, accg, udot, vsmx, diffu, hi4_i);
      }
    }
  }

  acc->x *= 0.5;
  acc->y *= 0.5;
  acc->z *= 0.5;
  accg->x *= 0.5;
  accg->y *= 0.5;
  accg->z *= 0.5;
  *udot *= 0.5;
  *diffu *= 2.;

}

__device__ void HydroForceKernel_2walk(struct Epj_HydroGPU (*jpsh)[N_THREAD_GPU],
				       const struct Epi_HydroGPU ip,
				       const Epj_HydroGPU *epj,
				       const int id_walk,
				       const int2 *ij_disp,
				       DF3 *acc,
				       DF3 *accg,
				       DF *udot,
				       DF *vsmx,
				       DF *diffu,
				       int iwalk0,
				       int iwalk1)
{
  const int jbeg0 = ij_disp[iwalk0].y;
  const int jbeg1 = ij_disp[iwalk1].y;
  const int jend0 = ij_disp[iwalk0+1].y;
  const int jend1 = ij_disp[iwalk1+1].y;
  const int nj0 = jend0 - jbeg0;
  const int nj1 = jend1 - jbeg1;

  const int nj_longer = nj0 > nj1 ? nj0 : nj1;
  const int nj_shorter = nj0 > nj1 ? nj1 : nj0;
  const int walk_longer = nj0 > nj1 ? 0 : 1;
  const int jbeg_longer = nj0 > nj1 ? jbeg0 : jbeg1;

  const int mywalk = id_walk==iwalk0 ? 0 : 1;

  const int tid = threadIdx.x;
 
  const DF hi4_i = ip.hinv * ip.hinv * ip.hinv * ip.hinv;

  for(int j=0; j<nj_shorter; j+=N_THREAD_GPU){
    jpsh[0][tid] = ((Epj_HydroGPU *)(epj + jbeg0 + j))[tid];
    jpsh[1][tid] = ((Epj_HydroGPU *)(epj + jbeg1 + j))[tid];

    if(nj_shorter-j < N_THREAD_GPU){
      for(int jj=0; jj<nj_shorter-j; jj++){	
	HydroForce_calc(ip, jpsh[mywalk][jj], acc, accg, udot, vsmx, diffu, hi4_i);
      }
    }else {
#pragma unroll
      for(int jj=0; jj<N_THREAD_GPU; jj++){
	HydroForce_calc(ip, jpsh[mywalk][jj], acc, accg, udot, vsmx, diffu, hi4_i);
      }
    }
  }

  for(int j=nj_shorter; j<nj_longer; j+=N_THREAD_GPU){
    jpsh[0][tid] = ((Epj_HydroGPU *)(epj + jbeg_longer + j))[tid];
    int jrem = nj_longer - j;
    if(jrem < N_THREAD_GPU){
      for(int jj=0; jj<jrem; jj++){
	if(mywalk == walk_longer)
	  HydroForce_calc(ip, jpsh[0][jj], acc, accg, udot, vsmx, diffu, hi4_i);
      }
    }else {
#pragma unroll
      for(int jj=0; jj<N_THREAD_GPU; jj++){
	if(mywalk == walk_longer)
	  HydroForce_calc(ip, jpsh[0][jj], acc, accg, udot, vsmx, diffu, hi4_i);
      }
    }
  }

  acc->x *= 0.5;
  acc->y *= 0.5;
  acc->z *= 0.5;
  accg->x *= 0.5;
  accg->y *= 0.5;
  accg->z *= 0.5;
  *udot *= 0.5;
  *diffu *= 2.;

}

__device__ void HydroForceKernel_multiwalk(const struct Epi_HydroGPU ip,
					   const Epj_HydroGPU *epj,
					   const int id_walk,
					   const int2 *ij_disp,
					   DF3 *acc,
					   DF3 *accg,
					   DF *udot,
					   DF *vsmx,
					   DF *diffu)
{ 
  const int j_head = ij_disp[id_walk  ].y;
  const int j_tail = ij_disp[id_walk+1].y;

  const DF hi4_i = ip.hinv * ip.hinv * ip.hinv * ip.hinv;
  for(int j=j_head; j<j_tail; j++){
    struct Epj_HydroGPU jp = epj[j];
    HydroForce_calc(ip, jp, acc, accg, udot, vsmx, diffu, hi4_i);
  }

  acc->x *= 0.5;
  acc->y *= 0.5;
  acc->z *= 0.5;
  accg->x *= 0.5;
  accg->y *= 0.5;
  accg->z *= 0.5;
  *udot *= 0.5;
  *diffu *= 2.;

}

__global__ void HydroForceKernel(const int2 *ij_disp,
				 const Epi_HydroGPU *epi,
				 const Epj_HydroGPU *epj,
				 HydroGPU *dev_hydro)
{
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  int t_head = blockDim.x * blockIdx.x;
  int t_tail = t_head + N_THREAD_GPU - 1;
  int nwalk_in_block = 1 + (epi[t_tail].id_walk - epi[t_head].id_walk);
#ifdef USE_FLOAT
  DF3 acc = make_float3(0.f, 0.f, 0.f);
  DF3 accg =  make_float3(0.f, 0.f, 0.f);
#endif
#ifdef USE_DOUBLE
  DF3 acc = make_double3(0.f, 0.f, 0.f);
  DF3 accg =  make_double3(0.f, 0.f, 0.f);
#endif

  DF udot=0.f;
  DF vsmx = 0.f;
  DF diffu = 0.f;

  int id_walk = epi[tid].id_walk;
  const struct Epi_HydroGPU ip = epi[tid];

  __shared__ struct Epj_HydroGPU jpsh[2][N_THREAD_GPU];
       
  if(1 == nwalk_in_block){
    HydroForceKernel_1walk(jpsh[0], ip, epj, id_walk, ij_disp, &acc, &accg, &udot, &vsmx, &diffu);       } else if(2 == nwalk_in_block){
    int iwalk0 = epi[t_head].id_walk;
    int iwalk1 = epi[t_tail].id_walk;
    HydroForceKernel_2walk(jpsh, ip, epj, id_walk, ij_disp, &acc, &accg, &udot, &vsmx, &diffu, iwalk0, iwalk1);
  } else{
    HydroForceKernel_multiwalk(ip, epj, id_walk, ij_disp, &acc, &accg, &udot, &vsmx, &diffu);
  }
  
  //HydroForceKernel_multiwalk(ip, epj, id_walk, ij_disp, &acc, &accg, &udot, &vsmx, &diffu);

  dev_hydro[tid].acc = acc;
  dev_hydro[tid].accg = accg;
  dev_hydro[tid].udot = udot;
  dev_hydro[tid].vsmx = vsmx;
  dev_hydro[tid].diffu = diffu;

}

static cudaPointer<Epi_HydroGPU>   dev_epi;
static cudaPointer<Epj_HydroGPU>   dev_epj;
static cudaPointer<HydroGPU> dev_hydro;
static cudaPointer<int2>     ij_disp;
static bool init_call = true;

PS::S32 DispatchKernel_HydroForce(const PS::S32 tag,
				  const PS::S32 n_walk,
				  const HydroEPI **epi,
				  const PS::S32 *nip,
				  const HydroEPJ **epj,
				  const PS::S32 *njp)
{
  assert(n_walk <= N_WALK_LIMIT);

  if(init_call){
    dev_epi   .allocate(NI_LIMIT);
    dev_epj   .allocate(NJ_LIMIT);
    dev_hydro .allocate(NI_LIMIT);
    ij_disp   .allocate(N_WALK_LIMIT+2);
    init_call = false;
  }

  ij_disp[0].x = 0;
  ij_disp[0].y = 0;
  for(int k=0; k<n_walk; k++){
    ij_disp[k+1].x = ij_disp[k].x + nip[k];
    ij_disp[k+1].y = ij_disp[k].y + njp[k];
  }
  ij_disp[n_walk+1] = ij_disp[n_walk];

  assert(ij_disp[n_walk].x < NI_LIMIT);
  assert(ij_disp[n_walk].y < NJ_LIMIT);
  ij_disp.htod(n_walk+2);

  int ni_tot_reg = ij_disp[n_walk].x;
  if(ni_tot_reg % N_THREAD_GPU){
    ni_tot_reg /= N_THREAD_GPU;
    ni_tot_reg++;
    ni_tot_reg *= N_THREAD_GPU;
  }

  int ni_tot = 0;
  int nj_tot = 0;

  int iw, i, j, num_i, num_j;
#pragma omp parallel for private(i,j, num_i, num_j) reduction(+:ni_tot, nj_tot)
  for(iw=0; iw<n_walk; iw++){
    for(i=0; i<nip[iw]; i++){
       num_i = ij_disp[iw].x + i;
      dev_epi[num_i].id = epi[iw][i].id;
      dev_epi[num_i].pos.x = epi[iw][i].pos.x;
      dev_epi[num_i].pos.y = epi[iw][i].pos.y;
      dev_epi[num_i].pos.z = epi[iw][i].pos.z;
      dev_epi[num_i].vel.x = epi[iw][i].vel.x;
      dev_epi[num_i].vel.y = epi[iw][i].vel.y;
      dev_epi[num_i].vel.z = epi[iw][i].vel.z;
      dev_epi[num_i].hinv = epi[iw][i].hinv;
      dev_epi[num_i].pres = epi[iw][i].pres;
      dev_epi[num_i].presc = epi[iw][i].presc;
      dev_epi[num_i].bswt = epi[iw][i].bswt;
      dev_epi[num_i].dens = epi[iw][i].dens;
      dev_epi[num_i].vsnd = epi[iw][i].vsnd;
      dev_epi[num_i].alph = epi[iw][i].alph;
      dev_epi[num_i].alphu = epi[iw][i].alphu;
      dev_epi[num_i].thrm = epi[iw][i].thrm;
      dev_epi[num_i].eta = epi[iw][i].eta;
      dev_epi[num_i].id_walk = iw;
      ni_tot++;
    }
    for(j=0; j<njp[iw]; j++){
      num_j = ij_disp[iw].y + j;
      dev_epj[num_j].id = epj[iw][j].id;
      dev_epj[num_j].pos.x = epj[iw][j].pos.x;
      dev_epj[num_j].pos.y = epj[iw][j].pos.y;
      dev_epj[num_j].pos.z = epj[iw][j].pos.z;
      dev_epj[num_j].vel.x = epj[iw][j].vel.x;
      dev_epj[num_j].vel.y = epj[iw][j].vel.y;
      dev_epj[num_j].vel.z = epj[iw][j].vel.z;
      dev_epj[num_j].hinv = epj[iw][j].hinv;
      dev_epj[num_j].pres = epj[iw][j].pres;
      dev_epj[num_j].presc = epj[iw][j].presc;
      dev_epj[num_j].bswt = epj[iw][j].bswt;
      dev_epj[num_j].dens = epj[iw][j].dens;
      dev_epj[num_j].vsnd = epj[iw][j].vsnd;
      dev_epj[num_j].alph = epj[iw][j].alph;
      dev_epj[num_j].alphu = epj[iw][j].alphu; 
      dev_epj[num_j].thrm = epj[iw][j].thrm;
      dev_epj[num_j].eta = epj[iw][j].eta;
      dev_epj[num_j].mass = epj[iw][j].mass;
      nj_tot++;
    }
  }

  for(int i=ni_tot; i<ni_tot_reg; i++){
    dev_epi[i].id_walk = n_walk;
  }

  dev_epi.htod(ni_tot_reg);
  dev_epj.htod(nj_tot);

  const DF ceff1 = +1.336901521971920914e+01;

  hipMemcpyToSymbol(HIP_SYMBOL(ceff1_c), &ceff1, sizeof(DF));

  int nblocks  = ni_tot_reg / N_THREAD_GPU;
  int nthreads = N_THREAD_GPU;
  
  // hipDeviceSetCacheConfig(hipFuncCachePreferEqual);  
  HydroForceKernel <<<nblocks, nthreads>>> (ij_disp, dev_epi, dev_epj, dev_hydro);

  return 0;
}

PS::S32 RetrieveKernel_HydroForce(const PS::S32 tag,
				  const PS::S32 n_walk,
				  const PS::S32 *ni,
				  Hydro         **hydro)
{
  int ni_tot = 0;
  for(int k=0; k<n_walk; k++){
    ni_tot += ni[k];
  }
  dev_hydro.dtoh(ni_tot);

  int n_cnt = 0;
  for(int iw=0; iw<n_walk; iw++){
    for(int i=0; i<ni[iw]; i++){ 
      hydro[iw][i].acch.x = dev_hydro[n_cnt].acc.x;
      hydro[iw][i].acch.y = dev_hydro[n_cnt].acc.y;
      hydro[iw][i].acch.z = dev_hydro[n_cnt].acc.z;
      hydro[iw][i].accg.x = dev_hydro[n_cnt].accg.x;
      hydro[iw][i].accg.y = dev_hydro[n_cnt].accg.y;
      hydro[iw][i].accg.z = dev_hydro[n_cnt].accg.z;
      hydro[iw][i].udot = dev_hydro[n_cnt].udot;
      hydro[iw][i].vsmx = dev_hydro[n_cnt].vsmx;
      hydro[iw][i].diffu = dev_hydro[n_cnt].diffu;
      n_cnt++;
    }
  }

  return 0;
}
