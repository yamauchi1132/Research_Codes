#include <stdio.h>
#include <hip/hip_runtime.h>
#include <particle_simulator.hpp>
#include "use_gpu.hpp"

void setup_gpu(int my_proc)
{
  if(my_proc == 0) {
    fprintf(stderr, "\n");
    fprintf(stderr, "###################\n");
    fprintf(stderr, "### Using GPU ! ###\n");
    fprintf(stderr, "###################\n");

    fprintf(stderr, "\n");
    fprintf(stderr, "Setting up GPU...\n");
  }

  //getting the number of GPU per node
  int ngpus;
  hipGetDeviceCount(&ngpus);

  //set up device for each process
  int dev = my_proc % ngpus;
  //fprintf(stderr, "%d %d %d test\n", dev, ngpus, my_proc);
  hipSetDevice(dev);

  //getting information of gpu
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev);
  if(my_proc == 0) {
    fprintf(stderr, "%d GPU per node available\n", ngpus);
    fprintf(stderr, "GPU name : %s\n\n", deviceProp.name);
  }
}

void reset_gpu(int my_proc)
{
  if(my_proc == 0) {
    fprintf(stderr, "\n");
    fprintf(stderr, "Resetting GPU...\n");
  }
  hipDeviceReset();
}
